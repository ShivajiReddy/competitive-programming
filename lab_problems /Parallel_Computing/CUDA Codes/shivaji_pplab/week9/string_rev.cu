#include "hip/hip_runtime.h"
// includes, system
#include <stdio.h>
#include <assert.h>

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char* msg);

// Part 1 of 1: implement the kernel
__global__ void reverseArrayBlock(int *d_b, int *d_a  )
{
 int bx= blockIdx.x ;
 int tx = threadIdx.x ;
 int i =  blockDim.x - 1 - (blockDim.x * bx + tx ) ;
 d_b[blockDim.x * bx + tx ] = d_a[i] ;



}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv)
{
    // pointer for host memory and size
    int *h_a;
    int dimA = 256;

    // pointer for device memory
    int *d_b, *d_a;

    // define grid and block size
    int numBlocks = 1;
    int numThreadsPerBlock = dimA;

    // allocate host and device memory
    size_t memSize = numBlocks * numThreadsPerBlock * sizeof(int);
    h_a = (int *) malloc(memSize);
    hipMalloc( (void **) &d_a, memSize );
    hipMalloc( (void **) &d_b, memSize );

    // Initialize input array on host
    for (int i = 0; i < dimA; ++i)
    {
        h_a[i] = i;
    }

    // Copy host array to device array
    hipMemcpy( d_a, h_a, memSize, hipMemcpyHostToDevice );

    // launch kernel
    dim3 dimGrid(numBlocks);
    dim3 dimBlock(numThreadsPerBlock);
    reverseArrayBlock<<< dimGrid, dimBlock >>>( d_b, d_a );

    // block until the device has completed
    hipDeviceSynchronize();

    // check if kernel execution generated an error
    // Check for any CUDA errors
    checkCUDAError("kernel invocation");

    // device to host copy
    hipMemcpy( h_a, d_b, memSize, hipMemcpyDeviceToHost );

    // Check for any CUDA errors
    checkCUDAError("memcpy");

    // verify the data returned to the host is correct
    for (int i = 0; i < dimA; i++)
    {
        assert(h_a[i] == dimA - 1 - i );
    }

    // free device memory
    hipFree(d_a);
    hipFree(d_b);

    // free host memory
    free(h_a);

    // If the program makes it this far, then the results are correct and
    // there are no run-time errors.  Good work!
    printf("Correct!\n");

    return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}