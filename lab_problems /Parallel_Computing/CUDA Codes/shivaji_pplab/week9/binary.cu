#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define N 1024

__global__ void binVal(char* A){

	int idx = threadIdx.x;
	int count = 0;
	int val = A[idx];
	int bin = 0;
	while(val>0){
		int rem = val%2;
		int c = count;
		while(c>0){
			rem *= 10;
			c--;
		}
		bin += rem;
		count++;
		val /= 2;
	}

	printf("%c - %d\n", A[idx], bin);
}

int main(){
	char A[N];
	char *pA;

	scanf("%s", A);
	int len = strlen(A);

	hipMalloc((void**)&pA, len*sizeof(char));
	hipMemcpy(pA, A, len*sizeof(char), hipMemcpyHostToDevice);
	hipError_t error =hipGetLastError();
	if (error != hipSuccess){
		printf("CUDA Error1: %s\n", hipGetErrorString(error));
	}
	binVal<<<1,len>>>(pA);
	error =hipGetLastError();

	if (error != hipSuccess){
		printf("CUDA Error2: %s\n", hipGetErrorString(error));
	}
	hipFree(pA);
	printf("\n");
	return 0;
}
