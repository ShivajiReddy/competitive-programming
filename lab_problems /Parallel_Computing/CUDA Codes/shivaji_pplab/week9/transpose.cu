#include "hip/hip_runtime.h"


#include <stdio.h>

//M x N
#define HA 2
#define WA 2
#define HB 2
#define WB 2

//rowwise
__global__ void transpose (int *A, int *B) {
    // Get the 1D Array index of the matrix
  int id=threadIdx.x;

  for(int i=0;i<HA;i++)
  {

	  B[i*WA+id] = A[id*HA+i];
  }
}

int main ()
{
    // Host copies of the variables
    int A[HA*WA], B[HA*WA];

    int i, j;
    for (i = 0; i < HA*WA; ++i) {
        A[i] = i + 1;
    }

    // Device copies of the variables
    int *d_a, *d_b;

    int size = sizeof(int)*HA*WA;

    // Allocate memories to device copies of the objects
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);

    // Copy inputs to device
    hipMemcpy(d_a, &A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &B, size, hipMemcpyHostToDevice);

    for (i = 0; i < HA; ++i) {
            for (j = 0; j < WA; ++j) {
                printf("%d\t", A[i * WA + j]);
            }
            printf("\n");
        }
    transpose<<<1, HA>>>(d_a, d_b);
    hipMemcpy(&B, d_b, size, hipMemcpyDeviceToHost);


    printf("\n");

    printf("B:\n");
    for (i = 0; i < HA; ++i) {
        for (j = 0; j < WA; ++j) {
            printf("%d\t", B[i * WA + j]);
        }
        printf("\n");
    }
    printf("\n");

    hipFree(d_a);
    hipFree(d_b);

    getchar();

    return 0;
}
