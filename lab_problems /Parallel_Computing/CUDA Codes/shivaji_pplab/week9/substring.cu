#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define N 1024

__global__ void findSubstr(char* A, char* C, int slen, int sublen){
	int idx = threadIdx.x;
	int flag = 1;
	for(int i=0;i<sublen;i++){
		if(A[idx+i] != C[i]){
			flag = 0;
			break;
		}
	}
	if(flag == 1){
		printf("\nSubstring found\n");
	}
	else
		printf("\nSubstring Not found!\n");
}
int main(){
	char A[N];
	char C[N];
	char *pA,*pC;
	scanf("%s", A);
	scanf("%s", C);
	/*cudaEvent_t start, stop;
	cudaEventCreate(&start);
	cudaEventCreate(&stop);
	cudaEventRecord(start, 0);*/

	int slen = strlen(A);
	int sublen = strlen(C);

	hipMalloc((void**)&pA, slen*sizeof(char));
	hipMalloc((void**)&pC, sublen*sizeof(char));
	hipMemcpy(pA, A, slen*sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(pC, C, sublen*sizeof(char), hipMemcpyHostToDevice);
	hipError_t error =hipGetLastError();
	if (error != hipSuccess){
		printf("CUDA Error1: %s\n", hipGetErrorString(error));
	}
	findSubstr<<<1,(slen-sublen+1)>>>(pA,pC, slen, sublen);
	error =hipGetLastError();

	if (error != hipSuccess){
		printf("CUDA Error2: %s\n", hipGetErrorString(error));
	}
	/*cudaEventRecord(stop, 0);
	cudaEventSynchronize(stop);
	float elapsedTime;
	cudaEventElapsedTime(&elapsedTime, start, stop);*/

	//printf("Time Taken=%f",elapsedTime);
	hipFree(pA);
	hipFree(pC);
	printf("\n");
	return 0;
}
