#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define WA  3
#define HA  3
#define WB  3
#define HB  3
#define WC  3
#define HC  3


__global__ void MatMult_one( int* A,  int* B, int* C)
{
		   int id=threadIdx.x;
		   int sum=0;

		   for(int w=0;w<WB;w++)
		   {
			   for(int z=0;z<HB;z++)

			   {
					sum += A[id*WA+z]*B[z*WB+w];
			   }
			   C[id*WA+w]=sum;

		   }
}

__global__ void MatMult_two( int* A,  int* B, int* C)
{

		   int id=threadIdx.x;
		   int sum=0;

		   for(int w=0;w<HA;w++)
		   {
			   for(int z=0;z<WA;z++)

			   {
					sum+=A[id*WA+z]*B[z*WB+w];
			   }
			   C[id*WA+w]=sum;

		   }


}

__device__ int getGlobalIdx_2D_1D(){
int blockIdYd = blockIdx.x * gridDim.y + blockIdx.y;
int threadIdYd = blockIdYd * blockDim.y + threadIdx.y;
return threadIdYd;
}

__global__ void MatMult_three( int* A,  int* B, int* C){

		   int idy=getGlobalIdx_2D_1D();


		  for(int i=0;i<gridDim.x;i++)
		  {
		   int id=idy*gridDim.x+i;


		   for(int w=0;w<WA;w++)
		   {
			   for(int z=0;z<HB;z++)

			   {
					C[w*WB+id]+= A[w*HB+z] * B[id+z*WB];
			   }

		   }
		  }

       }

int main(){

int A[WA*HA];
int B[WB*HB];
int C[WC*HC]={0};
printf("A: ");
for(int i=0;i<WA*HA;i++)
{
	scanf("%d", &A[i]);
}
printf("B: ");
for(int i=0;i<WB*HB;i++)
{
	scanf("%d", &B[i]);
}
int (*pA), (*pB), (*pC);

hipMalloc((void**)&pA, (WA*HA)*sizeof(int));
hipMalloc((void**)&pB, (WB*HB)*sizeof(int));
hipMalloc((void**)&pC, (WC*HC)*sizeof(int));

hipMemcpy(pA, A, (WA*HA)*sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(pB, B, (WB*HB)*sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(pC, C, (WC*HC)*sizeof(int), hipMemcpyHostToDevice);


MatMult_one<<<1,HA>>>(pA,pB,pC);

int choice = 0;
printf("Enter your choice: 1/2/3");
scanf("%d", &choice);
// Launch kernel onto the device
if(choice == 1)
{
	//rowwise
	MatMult_one<<<1,HA>>>(pA,pB,pC);
}
else if(choice == 2)
{
	//colwise
	MatMult_two<<<1,WB>>>(pA,pB,pC);

}
else
{
	//elementwise
	dim3 numBlocks = (3,3);
	dim3 threadsPerBlock(1,1);
	MatMult_three<<<numBlocks,threadsPerBlock>>>(pA,pB,pC);
}


hipMemcpy(C, pC, (WC*HC)*sizeof(int), hipMemcpyDeviceToHost);

int i, j=0;; printf("C = \n");
for(i=0;i<WC*HC;i++){
	if(j%WC==0) printf(" \n");
    printf("%d ", C[i]);
	j++;
    }
    printf("\n");


hipFree(pA);
hipFree(pB);
hipFree(pC);

printf("\n");

return 0;
}



