#include "hip/hip_runtime.h"


#include <stdio.h>

//M x N
#define M 3
#define N 4



// Each row of matrix is computed by one thread
__global__ void add_row (int *A, int *B, int *C) {
    // Get the 1D Array index of the matrix
    int idx = threadIdx.x;
    //printf("idx = %d\n", idx);
    for (int i = 0; i < M; ++i) {
        C[i + M * idx] = A[i + M * idx] + B[i + M * idx];
    }
}


// Each column is computed on one thread
__global__ void add_col (int *A, int *B, int *C) {
    // Get the 1D Array index of the matrix
    int idx = threadIdx.x;
    printf("idx = %d\n", idx);
    for (int i = 0; i < N; ++i, idx += M) {
        C[idx] = A[idx] + B[idx];
    }
}


// Each element is computed on one thread
__global__ void add_element (int *A, int *B, int *C) {
    // Get the 1D Array index of the matrix
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    printf("idx = %d\n", idx);
    C[idx] = A[idx] + B[idx];
}



int main () {

    // Host copies of the variables
    int A[M * N], B[M * N], C[M * N];

    int i, j;
    for (i = 0; i < M * N; ++i) {
        A[i] = i + 1;
        B[i] = M * N - i - 1;
    }

    // Device copies of the variables
    int *d_a, *d_b, *d_c;

    int size = sizeof(int) * M * N;

    // Allocate memories to device copies of the objects
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    // Copy inputs to device
    hipMemcpy(d_a, &A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &B, size, hipMemcpyHostToDevice);

    int choice = 0;
    printf("Enter your choice: 1/2/3");
    scanf("%d", &choice);
    // Launch kernel onto the device
    if(choice == 1)
    {
    	//rowwise
    	add_row<<<1, N>>>(d_a, d_b, d_c);
    }
    else if(choice == 2)
    {
    	//colwise
    	add_col<<<1, N>>>(d_a, d_b, d_c);

    }
    else
    {
    	//elementwise
        add_element<<<M, N>>>(d_a, d_b, d_c);
    }


    hipMemcpy(&C, d_c, size, hipMemcpyDeviceToHost);


    printf("A:\n");
    for (i = 0; i < N; ++i) {
        for (j = 0; j < M; ++j) {
            printf("%d\t", A[i * M + j]);
        }
        printf("\n");
    }
    printf("\n");

    printf("B:\n");
    for (i = 0; i < N; ++i) {
        for (j = 0; j < M; ++j) {
            printf("%d\t", B[i * M + j]);
        }
        printf("\n");
    }
    printf("\n");

    printf("C:\n");
    for (i = 0; i < N; ++i) {
        for (j = 0; j < M; ++j) {
            printf("%d\t", C[i * M + j]);
        }
        printf("\n");
    }
    printf("\n");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    getchar();

    return 0;
}
