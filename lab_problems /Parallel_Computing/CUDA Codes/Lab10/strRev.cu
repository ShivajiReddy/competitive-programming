
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void search(char *in, char* out){
	int i = threadIdx.x;
	int p = blockDim.x - threadIdx.x -1; 
	out[p] = in[i];
}
int main(){
	char *inp, *out;
	int n = 100;

	hipMallocManaged(&inp, n*sizeof(char)); 	
	hipMallocManaged(&out, n*sizeof(char)); 	

	//Taking input from user
	printf("Enter main string:\n");
	scanf("%s", inp);
	//Invoking the kernel
	out[strlen(inp)] = '\0';
	dim3 blockDim(strlen(inp));
	search<<<1,blockDim>>>(inp, out);
	hipDeviceSynchronize();


	//Printing the result	
	printf("Output string:%s\n", out);
	hipFree(inp);
	hipFree(out);
	return 0;
}
