
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void search(char *in, char* subs, int* found, int substrLen){
	int i = threadIdx.x;
	int start = i;
	for(int j = 0;j<substrLen;j++){
		if(in[start+j] != subs[j]){
			return;
		}
	}	
	found[0] = start;
}
int main(){
	char *inp, *substr;
	int *found;
	int n = 100;

	hipMallocManaged(&found, 1*sizeof(int)); 	
	hipMallocManaged(&inp, n*sizeof(char)); 	
	hipMallocManaged(&substr, n*sizeof(char)); 	

	//Taking input from user
	printf("Enter main string:\n");
	scanf("%s", inp);
	printf("Enter substring\n");
	scanf("%s", substr);
	found[0] = -1;

	//Invoking the kernel
	dim3 blockDim(strlen(inp) - strlen(substr) + 1);
	search<<<1,blockDim>>>(inp, substr, found, strlen(substr));
	hipDeviceSynchronize();

	//Printing the result	
	printf("Substring found at %d\n", found[0]);
	printf("\n");
	hipFree(found);
	hipFree(inp);
	hipFree(substr);
	return 0;
}
