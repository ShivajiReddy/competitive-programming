#include "hip/hip_runtime.h"
#include<stdio.h>
#include <string.h>
__global__ void search(char *inp, char* word, int* lengths, int *occ){
	int i = threadIdx.x;
	int len = lengths[i];
	int start = 0;
	for(int j = 0;j<i;j++){
		start+= lengths[j]; 
		start++;
	}
	printf("Thread %d start: %d\n", i, start);
	for(int j = 0;j<len;j++){
		if(inp[start+j] != word[j]){
			return;
		}
	}	
	printf("Found\n");
	atomicAdd(occ, 1);
}
int main(){
	char *inp, *substr;
	int *lengths;
	int *occ;
	int n = 100;
	int w = 50;
	hipMallocManaged(&lengths, w*sizeof(int)); 	
	hipMallocManaged(&occ, 1*sizeof(int)); 	
	hipMallocManaged(&inp, n*sizeof(char)); 	
	hipMallocManaged(&substr, n*sizeof(char)); 	

	//Taking input from user
	printf("Enter main string:\n");
	gets( inp);
	printf("Enter substring\n");
	scanf("%s", substr);
	
	char *p = strtok(inp, " ");
	char *words[w];
	occ[0] = 0;
	int i = 0;
	while (p != NULL){
		words[i] = p;
		lengths[i] = strlen(p);
		printf("Length %d is %d \n",i,  lengths[i]);
		i++;
		p = strtok(NULL, " ");
    	}
	//Invoking the kernel
	dim3 blockDim(i);
	search<<<1,blockDim>>>(inp, substr, lengths, occ);
	hipDeviceSynchronize();

	//Printing the result	
	printf("Occurences found:%d\n", occ[0]);
	hipFree(lengths);
	hipFree(inp);
	hipFree(substr);
	hipFree(occ);
	return 0;
}
