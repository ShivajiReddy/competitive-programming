#include "hip/hip_runtime.h"
#include<stdio.h>
#define ind(i, j, n) (i*n +j)
__global__ void add(int *a, int *b, int *c){
	int n = gridDim.x;
	int p = gridDim.y;
	int i = blockIdx.x;
	int j = blockIdx.y;
	c[ind(i, j, p)] = a[ind(i, j, p)]+b[ind(i, j, p)];
}

int main(){
	int *a, *b, *c;
	int n, p;
	printf("Enter N and P\n");
	scanf("%d %d", &n, &p);
	hipMallocManaged(&a, n*p*sizeof(int)); 	
	hipMallocManaged(&b, n*p*sizeof(int)); 	
	hipMallocManaged(&c, n*p*sizeof(int)); 	
	printf("Enter A\n");
	for(int i = 0;i<n;i++){
		for(int j = 0;j<p;j++){
			scanf("%d", &a[ind(i, j, p)]);
		}
	}
	printf("Enter B\n");
	for(int i = 0;i<n;i++){
		for(int j = 0;j<p;j++){
			scanf("%d", &b[ind(i, j, p)]);
		}
	}
	dim3 gridDim(n,p); 
	add<<<gridDim,1 >>>(a, b, c, n, p);
	hipDeviceSynchronize();
	printf(" Result \n");
	for(int i = 0;i<n;i++){
		for(int j = 0;j<p;j++){
			printf("%d ", c[ind(i, j, p)]);
		}
		printf("\n");
	}
	printf("\n");
	hipFree(a);
	hipFree(b);
	hipFree(c);
	return 0;
}
