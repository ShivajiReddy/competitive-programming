
#include <hip/hip_runtime.h>
#include<stdio.h>
#define ind(i, j, n) (i*n +j)
__global__ void add(int *a, int *b, int *c, int n, int p, int q){
	int i = blockIdx.x;
	int j = blockIdx.y;
	int k = 0;
	c[ind(i, j, q)] = 0;
	for(k=0;k<p;k++){
		c[ind(i,j,q)] += a[ind(i,k,p)] * b[ind(k,j,q)];	
	}
}

int main(){
	int *a, *b, *c;
	int n, p, q;
	printf("Enter N, P and Q\n");
	scanf("%d %d %d", &n, &p, &q);
	hipMallocManaged(&a, n*p*sizeof(int)); 	
	hipMallocManaged(&b, p*q*sizeof(int)); 	
	hipMallocManaged(&c, n*q*sizeof(int)); 	
	printf("Enter A\n");
	for(int i = 0;i<n;i++){
		for(int j = 0;j<p;j++){
			scanf("%d", &a[ind(i, j, p)]);
		}
	}
	printf("Enter B\n");
	for(int i = 0;i<p;i++){
		for(int j = 0;j<q;j++){
			scanf("%d", &b[ind(i, j, q)]);
		}
	}
	dim3 gridDim(n,q);
	add<<<gridDim,1>>>(a, b, c, n, p, q);
	hipDeviceSynchronize();
	printf(" Result \n");
	for(int i = 0;i<n;i++){
		for(int j = 0;j<q;j++){
			printf("%d ", c[ind(i, j, q)]);
		}
		printf("\n");
	}
	printf("\n");
	hipFree(a);
	hipFree(b);
	hipFree(c);
	return 0;
}
