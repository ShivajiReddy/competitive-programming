
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void toBin(char *in, int* out){
	int id = threadIdx.x;
	int binaryNum[1000]; 
	int n = in[id];
	int i = 0; 
	while (n > 0) { 
		// storing remainder in binary array 
		binaryNum[i] = n % 2; 
		n = n / 2; 
		i++; 
	} 
        // printing binary array in reverse order 
	int mult = 1;
	for(int j = i-1; j>0;j--){
		mult *= 10;
	}
	int output = 0;
	for (int j = i - 1; j >= 0; j--){
		output+= mult*binaryNum[j];
		mult/=10;
	}
	out[id] = output;
}

int main(){
	char *inp;
	int *out;
	int n = 100;

	hipMallocManaged(&inp, n*sizeof(char)); 	

	//Taking input from user
	printf("Enter string:\n");
	scanf("%s", inp);
	hipMallocManaged(&out, strlen(inp)*sizeof(int)); 	
	//Invoking the kernel
	dim3 blockDim(strlen(inp));
	toBin<<<1,blockDim>>>(inp, out);
	hipDeviceSynchronize();
	printf("%d\n", out[0]);
	//Printing the result	
	for(int i=0;i<strlen(inp);i++){
		printf("Char:%c(%d) --> %d\n", inp[i],inp[i],out[i]);
	}
	printf("\n");
	hipFree(inp);
	hipFree(out);
	return 0;
}
