#include "hip/hip_runtime.h"


#include <stdio.h>

// It's a M * N matrix
#define M 3
#define N 4

// Each row of matrix is computed by one thread

__global__ void mul (int *A, int *B, int *C) {
    // Get the 1D Array index of the matrix
    int idx = threadIdx.x;
    printf("idx = %d\n", idx);
    for (int i = 0; i < M; ++i) {
    	for (int j=0; j < N; ++j) {
        C[i + M * idx] += A[j + M * idx] * B[j * M + idx];
    	}
    }
}

int main () {

    // Host copies of the variables
    int A[M * N], B[M * N], C[M * N];

    int i, j;
    for (i = 0; i < M * N; ++i) {
        A[i] = i ;
        B[i] = i ;
    }

    // Device copies of the variables
    int *d_a, *d_b, *d_c;

    int size = sizeof(int) * M * N;

    // Allocate memories to device copies of the objects
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    // Copy inputs to device
    hipMemcpy(d_a, &A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &B, size, hipMemcpyHostToDevice);

    // Launch kernel onto the device
    mul<<<1,N>>>(d_a, d_b, d_c);

    // Copy the result back to the host
    hipMemcpy(&C, d_c, size, hipMemcpyDeviceToHost);

    // Outpoooot it
    printf("A:\n");
    for (i = 0; i < N; ++i) {
        for (j = 0; j < M; ++j) {
            printf("%d\t", A[i * M + j]);
        }
        printf("\n");
    }
    printf("\n");

    printf("B:\n");
    for (i = 0; i < M; ++i) {
        for (j = 0; j < N; ++j) {
            printf("%d\t", B[i * M + j]);
        }
        printf("\n");
    }
    printf("\n");

    printf("A * B:\n");
    for (i = 0; i < M; ++i) {
        for (j = 0; j < N; ++j) {
            printf("%d\t", C[i * M + j]);
        }
        printf("\n");
    }
    printf("\n");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);


    return 0;
}
