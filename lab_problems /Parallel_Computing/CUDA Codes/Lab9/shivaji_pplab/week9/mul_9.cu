#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define WA  9
#define HA  9
#define WB  9
#define HB  9
#define WC  9
#define HC  9


__global__ void MatMult_one( int* A,  int* B, int* C)
{
		   int id=threadIdx.x;


		   for(int w=0;w<WB;w++)
		   {
			   int sum=0;
			   for(int z=0;z<HB;z++)

			   {
					sum+=A[id*WA+z]*B[z*WB+w];
			   }
			   C[id*WA+w]=sum;

		   }
}

__global__ void MatMult_two( int* A,  int* B, int* C)
{

		   int id=threadIdx.x;


		   for(int w=0;w<HA;w++)
		   {
			   int sum=0;
			   for(int z=0;z<WA;z++)

			   {
					sum+=A[id*WA+z]*B[z*WB+w];
			   }
			   C[id*WA+w]=sum;

		   }


}

__device__ int getGlobalIdx_2D_1D(){
int blockIdYd = blockIdx.x * gridDim.y + blockIdx.y;
int threadIdYd = blockIdYd * blockDim.y + threadIdx.y;
return threadIdYd;
}

__global__ void MatMult_three( int* A,  int* B, int* C){

		   int idy=getGlobalIdx_2D_1D();


		  for(int i=0;i<gridDim.x;i++)
		  {
		   int id=idy*gridDim.x+i;


		   for(int w=0;w<WA;w++)
		   {
			   for(int z=0;z<HB;z++)

			   {
					C[w*WB+id]+= A[w*HB+z] * B[id+z*WB];
			   }

		   }
		  }

       }

int main(){

int A[WA*HA];
int B[WB*HB];
int C[WC*HC]={0};
for(int i=0;i<WA*HA;i++)
		A[i]=2;
for(int i=0;i<WB*HB;i++)
		B[i]=3;
int (*pA), (*pB), (*pC);

hipMalloc((void**)&pA, (WA*HA)*sizeof(int));
hipMalloc((void**)&pB, (WB*HB)*sizeof(int));
hipMalloc((void**)&pC, (WC*HC)*sizeof(int));

hipMemcpy(pA, A, (WA*HA)*sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(pB, B, (WB*HB)*sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(pC, C, (WC*HC)*sizeof(int), hipMemcpyHostToDevice);


MatMult_one<<<1,HA>>>(pA,pB,pC);

int choice = 0;
printf("Enter your choice: 1/2/3");
scanf("%d", &choice);
// Launch kernel onto the device
if(choice == 1)
{
	//rowwise
	MatMult_one<<<1,HA>>>(pA,pB,pC);
}
else if(choice == 2)
{
	//colwise
	MatMult_two<<<1,WB>>>(pA,pB,pC);

}
else
{
	//elementwise
	dim3 numBlocks = (3,3);
	dim3 threadsPerBlock(1,1);
	MatMult_three<<<numBlocks,threadsPerBlock>>>(pA,pB,pC);
}


hipMemcpy(C, pC, (WC*HC)*sizeof(int), hipMemcpyDeviceToHost);

int i, j=0;; printf("C = \n");
for(i=0;i<WC*HC;i++){
	if(j%WC==0) printf(" \n");
    printf("%d ", C[i]);
	j++;
    }
    printf("\n");


hipFree(pA);
hipFree(pB);
hipFree(pC);

printf("\n");

return 0;
}


