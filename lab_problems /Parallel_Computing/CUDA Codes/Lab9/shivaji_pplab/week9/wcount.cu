#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define N 9

__device__ int getGlobalIdx_1D_1D(){
return blockIdx.x *blockDim.x + threadIdx.x;
}

__global__ void find( char* A, char* B, int* C, int len){
int id=getGlobalIdx_1D_1D();
int flag=0;
for(int i=0;i<len;i++){
if(A[id+i]!=B[i]){
flag=1;
}
}
if(flag==0){
C[id]=1;
}
else{
C[id]=0;
}
}

int main(){
char A[1000];
char B[1000];
int C[1000]={0};
char ca;
printf("Enter the sentence\n");
gets(A);
scanf("%c",&ca);
printf("Enter the word\n");
gets(B);
int len1 = strlen(A);
int len2 = strlen(B);
char (*pA), (*pB);
int (*pC);

hipMalloc((void**)&pA, (len1)*sizeof(char));
hipMalloc((void**)&pB, (len2)*sizeof(char));
hipMalloc((void**)&pC, (len1-len2+1)*sizeof(int));

hipMemcpy(pA, A, (len1)*sizeof(char), hipMemcpyHostToDevice);
hipMemcpy(pB, B, (len2)*sizeof(char), hipMemcpyHostToDevice);

int numBlocks = 1;
int threadsPerBlock = len1-len2+1;
find<<<numBlocks,threadsPerBlock>>>(pA,pB,pC,len2);

hipMemcpy(C, pC, (len1-len2+1)*sizeof(int), hipMemcpyDeviceToHost);

int i,sum=0;
printf("Number of times given word is in sentence = \n");
for(i=0;i<len1-len2+1;i++){
sum+=C[i];
}
printf("%d\n",sum);

hipFree(pA);
hipFree(pB);
hipFree(pC);
printf("\n");
return 0;
}



