#include "hip/hip_runtime.h"


#include <stdio.h>

// It's a M * N matrix
#define M 3
#define N 4

// Each column is computed on one thread

__global__ void add (int *A, int *B, int *C) {
    // Get the 1D Array index of the matrix
    int idx = threadIdx.x;
    printf("idx = %d\n", idx);
    for (int i = 0; i < N; ++i, idx += M) {
        C[idx] = A[idx] + B[idx];
    }
}

int main () {

    // Host copies of the variables
    int A[M * N], B[M * N], C[M * N];

    int i, j;
    for (i = 0; i < M * N; ++i) {
        A[i] = i + 1;
        B[i] = M * N - i - 1;
    }

    // Device copies of the variables
    int *d_a, *d_b, *d_c;

    int size = sizeof(int) * M * N;

    // Allocate memories to device copies of the objects
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    // Copy inputs to device
    hipMemcpy(d_a, &A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &B, size, hipMemcpyHostToDevice);

    // Launch kernel onto the device
    add<<<1, N>>>(d_a, d_b, d_c);

    // Copy the result back to the host
    hipMemcpy(&C, d_c, size, hipMemcpyDeviceToHost);

    // Outpoooot it
    printf("A:\n");
    for (i = 0; i < N; ++i) {
        for (j = 0; j < M; ++j) {
            printf("%d\t", A[i * M + j]);
        }
        printf("\n");
    }
    printf("\n");

    printf("B:\n");
    for (i = 0; i < N; ++i) {
        for (j = 0; j < M; ++j) {
            printf("%d\t", B[i * M + j]);
        }
        printf("\n");
    }
    printf("\n");

    printf("A + B:\n");
    for (i = 0; i < N; ++i) {
        for (j = 0; j < M; ++j) {
            printf("%d\t", C[i * M + j]);
        }
        printf("\n");
    }
    printf("\n");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);


    return 0;
}
